#include "hip/hip_runtime.h"
// Также в примере будет продемонстрирован расчет эффективной
// пропускной способности и как ее считать.

#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include <math.h>
#include <stdlib.h>

__global__ void saxpy(int n, float a, float *x, float *y)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n) {
		y[i] = a*x[i] + y[i];
	}
}

int main()
{
	int N = 1<<23;
	int size = N*sizeof(float);
	printf("N = %d\n", N);

	// Создание обработчиков событий CUDA
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	float *x, *y; // Вектора на хосте
	float *d_x, *d_y; // Вектора на устройстве
	// Выделение памяти на хосте
	x = (float *)malloc(size);
	y = (float *)malloc(size);
	// Выделение памяти на устройстве
	hipMalloc(&d_x, size);
	hipMalloc(&d_y, size);

	for (int i = 0; i < N; ++i)
	{
		x[i] = 1.0f;
		y[i] = 2.0f;
	}

	hipMemcpy(d_x, x, size, hipMemcpyHostToDevice);
	hipMemcpy(d_y, y, size, hipMemcpyHostToDevice);
	// Выполнение SAXPY на 1 миллионе элементов
	hipEventRecord(start);
	saxpy<<<(N+255)/256, 256>>>(N, 2.0, d_x, d_y);
	hipEventRecord(stop);

	hipMemcpy(y, d_y, size, hipMemcpyDeviceToHost);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	float maxError = 0.0f;
	for (int i = 0; i < N; ++i)
	{
		maxError = max(maxError, abs(y[i]-4.0f));
	}
	printf("Max error: %f\n", maxError);
	//Время выполнения программы на N элементах
	printf("Succesfully performed SAXPY on %d elements in %f milliseconds.\n", 
			N, milliseconds);
	//Эффективная пропускная способность Гб/сек
	printf("Effective Bandwidth (GB/s): %f\n", N*4*3/milliseconds/1e6);

	return 0;
}