#include "hip/hip_runtime.h"
// Сложение двух чисел с использованием N-блоков по одному треду

__global__ void add(int *a, int *b, int *c) 
{
	c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

#define N 512

int main(void) 
{
	 // host копии a, b, c
	int *a, *b, *c;
	// device копии a, b, c
	int *dev_a, *dev_b, *dev_c; 
	
	int size = N * sizeof(int);

	//выделяем память для device копий a, b, c
	hipMalloc((void**)&dev_a, size);
	hipMalloc((void**)&dev_b, size);
	hipMalloc((void**)&dev_c, size);

	a = (int*)malloc(size);
	b = (int*)malloc(size);
	c = (int*)malloc(size);

	random_ints(a, N);
	random_ints(b, N);

	// копируем ввод на device
	hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, size, hipMemcpyHostToDevice);
	
	// launch add() kernel with N parallel blocks
	add<<< N, 1 >>>(dev_a, dev_b, dev_c);

	// копируем результат работы device обратно на host – копию c
	hipMemcpy(c, dev_c, size, hipMemcpyDeviceToHost);

	free(a); free(b); free(c);
	hipFree(dev_a); hipFree(dev_b); hipFree(dev_c);
	
	return 0;
}