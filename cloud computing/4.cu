#include "hip/hip_runtime.h"
// Сложение двух чисел с одним блоком и N-тредов

__global__ void add(int *a, int *b, int *c) 
{
	c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];
	// blockIdx.x blockIdx.x blockIdx.x
}

#define N 512

int main(void) 
{
	//host копии a, b, c
	int *a, *b, *c; 
	//device копии of a, b, c
	int *dev_a, *dev_b, *dev_c;
	
	int size = N * sizeof(int);
	//выделяем память для копий a, b, c
	hipMalloc((void**)&dev_a, size);
	hipMalloc((void**)&dev_b, size);
	hipMalloc((void**)&dev_c, size);

	a = (int*)malloc(size);
	b = (int*)malloc(size);
	c = (int*)malloc(size);
	random_ints(a, N);
	random_ints(b, N);

	// копируем ввод device
	hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, size, hipMemcpyHostToDevice);
	
	// запускаем на выполнение add() kernel с N тредами в блоке
	add<<< 1, N >>>(dev_a, dev_b, dev_c);

	// копируем результат работы device обратно на host (копия c)
	hipMemcpy(c, dev_c, size, hipMemcpyDeviceToHost);

	free(a); free(b); free(c);
	hipFree(dev_a); hipFree(dev_b); hipFree(dev_c);

	return 0;
}