// Сложение двух векторов 

#include <cstdlib>
#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;

__global__ 
void vecAdd_kernel(const float *a, const float *b, float *result, int n) 
{
	// обозначающую размерность блоков и грида - одномерная
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n)
		result[i] = a[i] + b[i];
}

int main() 
{	
	int n = 100;
	float *a = new float[n], *a_gpu;
	hipMalloc((void**)&a_gpu, n*sizeof(float));

	float *b = new float[n], *b_gpu;
	hipMalloc((void**)&b_gpu, n*sizeof(float));

	float *result = new float[n], *result_gpu;
	hipMalloc((void**)&result_gpu, n*sizeof(float));

	for (size_t i = 0; i < n; ++i)
		a[i] = b[i] = i;

	// Далее происходит копирование данных из памяти CPU в память GPU
	hipMemcpy(a_gpu, a, n*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(b_gpu, b, n*sizeof(float), hipMemcpyHostToDevice);

	const int block_size = 256;
	int num_blockds = (n + block_size - 1) / block_size;

	vecAdd_kernel<<< num_blockds, block_size >>> (a_gpu, b_gpu, result_gpu, n);

	hipMemcpy(result, result_gpu, n*sizeof(float), hipMemcpyDeviceToHost);

	for (size_t x = 0; x < 10; ++x)	
		cout << result[x] << endl;


	delete[] a, delete[] b, delete[] result;
	hipFree(a_gpu); hipFree(b_gpu); hipFree(result_gpu);

	return 0; 
}