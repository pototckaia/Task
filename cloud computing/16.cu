#include "hip/hip_runtime.h"
//Реализация фрактала Множество Мандельброта
// Библиотеки CUDA и OpenCV
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <string>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>

using namespace cv;
using namespace std;

#define HEIGHT 512 // Кратность block_size.y
#define WIDTH 512 // Кратность block_size.x
#define MAX_ITER 10000

// Функция множества Мандельброта
void mandelbrotGPU(uchar*);
__global__ void calc(char* image_buffer);


#define cudaAssertSuccess(ans)\
{ _cudaAssertSuccess((ans), __FILE__, __LINE__); }

// Функция Assert’а
inline void _cudaAssertSuccess(hipError_t code, char *file, int line) {
    if (code != hipSuccess) {
        fprintf(stderr,"_cudaAssertSuccess: %s %s %d\n", hipGetErrorString(code), file, line);
        exit(code);
    }
}

int main(int argc, char** argv) {
    // Рендеринг множества
    cv::Mat image_output = cv::Mat(WIDTH, HEIGHT, CV_8UC1);
    // image* image_output = cvCreateImage(cvSize(WIDTH, HEIGHT), IPL_DEPTH_8U,1);
    mandelbrotGPU(image_output.data);
    cv::namedWindow("GPU", 256);
    cv::imshow("GPU", image_output);
    cv::waitKey();
}

// Работа GPU с множеством Мандельброта
void mandelbrotGPU(uchar* image_buffer) {
    char* d_image_buffer;
    cudaAssertSuccess(hipMalloc(&d_image_buffer, WIDTH * HEIGHT));
    dim3 block_size(16, 16);
    dim3 grid_size(WIDTH / block_size.x, HEIGHT / block_size.y);
    calc<<<grid_size, block_size>>>(d_image_buffer);
    cudaAssertSuccess(hipPeekAtLastError());
    cudaAssertSuccess(hipDeviceSynchronize());
    cudaAssertSuccess(hipMemcpy(image_buffer, d_image_buffer, HEIGHT * WIDTH,
    hipMemcpyDeviceToHost));
    cudaAssertSuccess(hipFree(d_image_buffer));
}

__global__ void calc(char* image_buffer) {
    int row = blockIdx.y * blockDim.y + threadIdx.y; // WIDTH
    int col = blockIdx.x * blockDim.x + threadIdx.x; // HEIGHT
    int idx = row * WIDTH + col;
    if(col >= WIDTH || row >= HEIGHT) 
        return;
    float x0 = ((float)col / WIDTH) * 3.5f - 2.5f;
    float y0 = ((float)row / HEIGHT) * 3.5f - 1.75f;
    float x = 0.0f;
    float y = 0.0f;
    int iter = 0;
    float xtemp;
    // Основная часть построения множества Мандельброта
    while((x * x + y * y <= 4.0f) && (iter < MAX_ITER)) {
        xtemp = x * x - y * y + x0;
        y = 2.0f * x * y + y0;
        x = xtemp;
        iter++;
    }

    // Цвет рендеринга
    int color = iter * 5;
    if (color >= 256) color = 0;
    image_buffer[idx] = color;
}