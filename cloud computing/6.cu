#include "hip/hip_runtime.h"
// Скалярное произведение с использованием shared-памяти 

#define N 512

__global__ void dot(int *a, int *b, int *c) 
{
	__shared__ int temp[N];
	temp[threadIdx.x] = a[threadIdx.x] * b[threadIdx.x];
	__syncthreads();
	if (0 == threadIdx.x) 
	{
		int sum = 0;
		for(int i = 0; i < N; ++i)
			sum += temp[i];
		*c = sum;
	}
}

int main(void) 
{
	int *a, *b, *c;
	int *dev_a, *dev_b, *dev_c;
	
	int size = N * sizeof( int );
	hipMalloc( (void**)&dev_a, size );
	hipMalloc( (void**)&dev_b, size );
	hipMalloc( (void**)&dev_c, sizeof( int ) );

	a = (int *)malloc( size );
	b = (int *)malloc( size );
	c = (int *)malloc( sizeof( int ) );

	random_ints( a, N );
	random_ints( b, N );

	// копируем ввод на device
	hipMemcpy( dev_a, a, size, hipMemcpyHostToDevice );
	hipMemcpy( dev_b, b, size, hipMemcpyHostToDevice );

	//запускаем на выполнение dot() kernel с 1 блоком и N тредами
	dot<<< 1, N >>>( dev_a, dev_b, dev_c );

	//копируем результат работы device на host копией c
	hipMemcpy( c, dev_c, sizeof( int ) , hipMemcpyDeviceToHost );

	free( a ); free( b ); free( c );
	hipFree( dev_a ); hipFree( dev_b ); hipFree( dev_c );

	return 0;
}