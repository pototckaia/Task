// Сложение векторов и сравнение с количеством тредов в памяти

#include <iostream>

#include <hip/hip_runtime.h>

using namespace std;

__global__ void add(float *a, float *b, float *c)
{
	if (a[threadIdx.x] + b[threadIdx.x] < 10)
		c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];
	else
		c[threadIdx.x] = 10;
}

//Количество суммирований и сравнений
#define N 64

int main(void) 
{
	float *a, *b, *c; // host копии a, b, c
	float *dev_a, *dev_b, *dev_c; // device копии a, b, c
	int size = N * sizeof(float);

	//выделяем память для device копий a, b, c
	hipMalloc((void**)&dev_a, size);
	hipMalloc((void**)&dev_b, size);
	hipMalloc((void**)&dev_c, size);
	a = (float*) malloc(size);
	b = (float*) malloc(size);
	c = (float*) malloc(size);
	for (int i = 0; i < N; ++i)
		a[i] = (float)rand() / (float)RAND_MAX;
	for (int i = 0; i < N; ++i)
		b[i] = (float)rand() / (float)RAND_MAX;

	// копируем ввод на device
	hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, size, hipMemcpyHostToDevice);
	// launch add() kernel with N parallel blocks
	//Использование N тредов и 1 блока
	add<<< 1, N >>>(dev_a, dev_b, dev_c);
	//Использование N блоков И 1 тре-
	// Если мы будем использовать N блоков и 1 тред 
	// add<<< N, 1 >>>(dev_a, dev_b, dev_c);
	// то каждый результат с будет равным нулю, кроме первого
	//add<<< N, 1 >>>(dev_a, dev_b, dev_c);

	// копируем результат работы device обратно на host – копию c
	hipMemcpy(c, dev_c, size, hipMemcpyDeviceToHost);

	for(int i = 0; i < N; ++i)
		cout<<"c[" << i << "]=" << c[i] << endl;

	free(a); free(b); free(c);
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	return 0;
}