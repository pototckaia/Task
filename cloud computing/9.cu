// Перемножение двух матриц


#include <hip/hip_runtime.h>
#include <stdio.h>

#define BLOCK_SIZE 16 // submatrix size
#define N 1024 // matrix size is N*N

__global__ void matMult (float *a, float *b, int n, float *c)
{
	int bx = blockIdx.x;
	// block index
	int by = blockIdx.y;
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	float sum = 0.0f;
	// computed subelement
	// a [i][0]
	int ia = n * BLOCK_SIZE * by + n * ty;
	int ib = BLOCK_SIZE * bx + tx;
	// Multiply the two matrices together;
	for (int k = 0; k < n; ++k)
		sum += a[ia + k] * b[ib + k*n];
	// Write the block sub-matrix to global memory;
	// each thread writes one element
	int ic = n * BLOCK_SIZE * by + BLOCK_SIZE * bx;

	c[ic + n * ty + tx] = sum;
}

int main (int argc, char *argv[])
{
	int numBytes = N * N * sizeof(float);
	
	// выделение памяти на хосте
	float *a = new float [N*N];
	float *b = new float [N*N];
	float *c = new float [N*N];
	
	for (int i = 0; i < N; ++i)
		for ( int j = 0; j < N; ++j)
		{
			int k = N*i + j;
			a[k] = 0.0f;
			b[k] = 1.0f;
		}
	
	// выделение памяти на девайсе
	float *adev = NULL;
	float *bdev = NULL;
	float *cdev = NULL;
	hipMalloc((void**)&adev, numBytes);
	hipMalloc((void**)&cdev, numBytes);
	hipMalloc((void**)&bdev, numBytes);

	// Установка конфигурации запуска ядра
	dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
	dim3 blocks(N / threads.x, N / threads.y);
	
	// Создание обработчика событий CUDA
	hipEvent_t start, stop;
	float gpuTime = 0.0f;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// асинхронно выдаваем работу на GPU (все в поток 0)
	hipEventRecord(start, 0 );
	hipMemcpy(adev, a, numBytes, hipMemcpyHostToDevice);
	hipMemcpy(bdev, b, numBytes, hipMemcpyHostToDevice);
	matMult<<<blocks, threads>>>(adev, bdev, N, cdev);
	hipMemcpy(c, cdev, numBytes, hipMemcpyDeviceToHost );
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&gpuTime, start, stop);
	
	// Печатаем время работы на GPU и CPU
	printf("time spent executing by the GPU: %.2f millseconds\n", gpuTime);
	
	// Освобождение ресурсов
	hipEventDestroy (start); hipEventDestroy(stop);
	hipFree(adev); hipFree(bdev); hipFree(cdev);

	delete a; delete b; delete c;
	
	return 0;
}