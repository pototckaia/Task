#include "hip/hip_runtime.h"
// Увеличение значений элементов в матрице на единицу

#include <stdio.h>

// каждому треду соответствует один тред, блоки и grid одномерны. 
// Ядро (она же функция incKernel) на вход получает только указатель на массив 
// с данными в глобальной памяти. 
// Задача ядра – по threadIdx и blockIdx определить, какой именно элемент 
// соответствует данному треду, и увеличить именно его.
__global__ void incKernel (float *data)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	data[idx] = data[idx] + 1.0f;
}

int main (int argc, char *argv[])
{
	int n = 16 * 1024 * 1024;
	int numBytes = n * sizeof(float);

	// выделение памяти на хосте
	float *a = new float[n];
	for (sts::size_t i = 0; i < n; ++i)
		a[i] = 0.0f;

	// выделение памяти на девайсе
	float *dev = NULL;
	hipMalloc ((void**)&dev, numBytes);

	// Устоновка конфигурации запуска ядра
	dim3 threads = dim3(512, 1);
	dim3 blocks = dim3(n / threads.x, 1);

	// создание обработчиков событий cuda
	hipEvent_t start, stop;
	float gpuTime = 0.0f;

	hipEventCreate(&start);
	hipEventCreate(&stop);

	// асинхронно выдаем работу на GPU (все в поток 0)
	hipEventRecord(start, 0);
	hipMemcpy(dev, a, numBytes, hipMemcpyHostToDevice);

	incKernel<<<blocks, threads>>>(dev);

	hipMemcpy(a, dev, numBytes, hipMemcpyDeviceToHost);
	hipEventRecord(stop, 0);

	hipEventSynchronize(stop);
	hipEventElapsedTime(&gpuTime, start, stop);

	// Печатаем время работы на CPU и GPU
	printf("time spent executing by the GPU: %.2f millseconds\n", 
			gpuTime);

	// проверка аутпута на корректность
	printf("--------------------------------------------------------------\n");
	for (size_t i = 0; i < n; ++i)
		if (a[i] != 1.0f) 
		{
			printf("Error in pos %d, %f\n", i, a[i]);
			break;
		}


	hipEventDestroy(start); hipEventDestroy(stop);
	hipFree(dev);
	delete a;

	return 0;
}