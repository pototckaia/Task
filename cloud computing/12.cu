#include "hip/hip_runtime.h"
// Генерация псевдослучайных чисел с использованием CuRand

#include <stdio.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define MAX 100

/*
эта функция ядра GPU вычисляет случайное число 
и сохраняет его в памяти
*/
__global__ void random(unsigned int seed, int *result) 
{
	/* 
	Библиотека случайных чисел CUDA использует hiprandState_t 
	для отслеживания начального значения
	мы будем хранить случайное состояние для каждого потока
	*/
	hiprandState_t state;
	/* инициализация состояния*/
	/* seed контролирует последовательность значений, которые
	генерируются*/
	/* порядковый номер важен только с несколькими ядрами*/
	hiprand_init(seed, 0, 0, &state);

	/* hiprand работает как rand – 
	за исключением того, что он принимает состояние как параметр*/
	*result = hiprand(&state) % MAX;
}

int main() 
{
	/* выделить память int на GPU*/
	int *gpu_x;
	hipMalloc((void**) &gpu_x, sizeof(int));

	/* вызывать GPU для инициализации всех случайных состояний*/
	random<<<1, 1>>>(time(NULL), gpu_x);
	/* скопировать случайное число на CPU*/

	int x;
	hipMemcpy(&x, gpu_x, sizeof(int), hipMemcpyDeviceToHost);

	printf("Random number = %d.\n", x);

	/* освобождение памяти */
	hipFree(gpu_x);

	return 0;
}