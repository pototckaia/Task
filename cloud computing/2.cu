
#include <hip/hip_runtime.h>
#include<iostream>

__global__ void add(int *a, int *b, int *c )
{
	*c = *a+*b;
}

int main(void )
{
	int a, b, c;
	int *dev_a, *dev_b, *dev_c;
	int size = sizeof( int );
	hipMalloc( (void**)&dev_a, size );
	hipMalloc( (void**)&dev_b, size );
	hipMalloc( (void**)&dev_c, size );
	a = 5;
	b = 10;
	hipMemcpy(dev_a, &a,size,hipMemcpyHostToDevice);
	hipMemcpy(dev_b, &b, size, hipMemcpyHostToDevice);
	
	add<<<1,1>>>(dev_a, dev_b, dev_c);
	hipMemcpy(&c,dev_c,size,hipMemcpyDeviceToHost);
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	std::cout<<a<<" "<<b<<" "<<c<<std::endl;
	return 0;
}
