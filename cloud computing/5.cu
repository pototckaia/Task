// Сложение двух чисел с использованием N-блоков и N-тредов



#include <hip/hip_runtime.h>
#include <iostream>

void random_ints(int *a, int n)
{
   int i;
   for (i = 0; i < n; ++i)
       a[i] = rand() %5000;
}

__global__ void add(int *a, int *b, int *c) 
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	c[index] = a[index] + b[index];
}

#define N (2048*2048)
#define THREADS_PER_BLOCK 512

int main(void) 
{
	// host копии a, b, c
	int *a, *b, *c; 
	// device копии of a, b, c
	int *dev_a, *dev_b, *dev_c; 
	
	int size = N * sizeof(int);
	//выделяем память на device для of a, b, c
	hipMalloc((void**)&dev_a, size);
	hipMalloc((void**)&dev_b, size);
	hipMalloc((void**)&dev_c, size);
	
	a = (int*)malloc(size);
	b = (int*)malloc(size);
	c = (int*)malloc(size);

	random_ints(a, N);
	random_ints(b, N);

	//копируем ввод на device
	hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, size, hipMemcpyHostToDevice);

	//запускаем на выполнение add() kernel с блоками и тредами
	add<<< N/THREADS_PER_BLOCK, THREADS_PER_BLOCK >>>(dev_a, dev_b, dev_c);
	
	// копируем результат работы device на host ( копия c )
	hipMemcpy(c, dev_c, size, hipMemcpyDeviceToHost);

	
	for (std::size_t i = 0; i < 10; ++i) {
		std::cout << a[i] << "+" << b[i] << "=" << c[i] << std::endl; 
	}

	free(a); free(b); free(c);
	hipFree(dev_a); hipFree(dev_b); hipFree(dev_c);

	return 0;
}