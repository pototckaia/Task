// Перемножение двух матриц с использованием shared-памяти


#include <hip/hip_runtime.h>
#include <stdio.h>

#define BLOCK_SIZE 16 // submatrix size
#define N 1024 // matrix size is N*N

__global__ void matMult(float *a, float *b, int n, float *c)
{
	int bx = blockIdx.x;
	int by = blockIdx.y;
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	// Index of the first sub-matrix of A processed by the block
	int aBegin = n * BLOCK_SIZE * by;
	int aEnd = aBegin + n - 1;

	// Step size used to iterate through the sub-matrices of A
	int aStep = BLOCK_SIZE;

	// Index of the first sub-matrix of B processed by the block
	int bBegin = BLOCK_SIZE * bx;

	// Step size used to iterate through the sub-matrices of B
	int bStep = BLOCK_SIZE * n;
	
	float sum = 0.0f;
	// computed subelement
	for(int ia = aBegin, ib = bBegin; ia <= aEnd; ia += aStep, ib += bStep)
	{
		// Shared memory for the sub-matrix of A
		__shared__ float as [BLOCK_SIZE][BLOCK_SIZE];
		// Shared memory for the sub-matrix of B
		__shared__ float bs [BLOCK_SIZE][BLOCK_SIZE];
		// Load the matrices from global memory to shared memory;
		as[ty][tx] = a[ia + n * ty + tx];
		bs[ty][tx] = b[ib + n * ty + tx];
		// Synchronize to make sure the matrices are loaded
		__syncthreads();
		// Multiply the two matrices together;
		for(int k = 0; k < BLOCK_SIZE; k++)
			sum += as [ty][k] * bs [k][tx];
		// Synchronize to make sure that the preceding
		// computation is done before loading two new
		// sub-matrices of A and B in the next iteration
		__syncthreads();
	}
	// Write the block sub-matrix to global memory;
	// each thread writes one element
	int ic = n * BLOCK_SIZE * by + BLOCK_SIZE * bx;
	c [ic + n * ty + tx] = sum;
}

int main(int argc, char *argv[])
{
	int numBytes = N * N * sizeof(float);

	// allocate host memory
	float *a = new float [N*N];
	float *b = new float [N*N];
	float *c = new float [N*N];
	for(int i = 0; i < N; ++i)
		for(int j = 0; j < N; ++j)
		{
			a [i] = 0.0f;
			b [i] = 1.0f;
		}

	// allocate device memory
	float *adev = NULL;
	float *bdev = NULL;
	float *cdev = NULL;
	hipMalloc((void**)&adev, numBytes);
	hipMalloc((void**)&bdev, numBytes);
	hipMalloc((void**)&cdev, numBytes);
	
	// set kernel launch configuration
	dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
	dim3 blocks(N / threads.x, N / threads.y);
	
	// create cuda event handles
	hipEvent_t start, stop;
	float gpuTime = 0.0f;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// asynchronously issue work to the GPU (all to stream 0)
	hipEventRecord(start, 0);
	hipMemcpy(adev, a, numBytes, hipMemcpyHostToDevice);
	hipMemcpy(bdev, b, numBytes, hipMemcpyHostToDevice);
	matMult<<<blocks, threads>>>(adev, bdev, N, cdev);
	hipMemcpy(c, cdev, numBytes, hipMemcpyDeviceToHost);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&gpuTime, start, stop);

	// print the cpu and gpu times
	printf("time spent executing by the GPU: %.2f millseconds\n", gpuTime);

	// release resources

	hipEventDestroy(start); hipEventDestroy(stop);
	hipFree(adev); hipFree(bdev); hipFree(cdev);
	delete a; delete b; delete c;

	return 0;
}