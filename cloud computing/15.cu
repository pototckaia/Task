
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>

/* 
Каждый поток получает ровно одно значение 
в несортированном массиве 
*/
#define THREADS 512 // 2^9
#define BLOCKS 32768 // 2^15
#define NUM_VALS (THREADS*BLOCKS)

/* 
Функция печати результата выполнения программы
*/
void print_elapsed(clock_t start, clock_t stop)
{
	double elapsed = ((double) (stop - start)) / CLOCKS_PER_SEC;
	printf("Elapsed time: %.3fs\n", elapsed);
}

/* Генерация случайных чисел массива */
float random_float()
{
	return (float)rand()/(float)RAND_MAX;
}

/* Печать массива */
void array_print(float *arr [], int length)
{
	for (int i = 0; i < length; ++i) {
		printf("%1.3f", arr[i]);
	}
	printf("\n");
}

/* Заполнение массива */
void array_fill(float *arr, int length)
{
	srand(time(NULL));
	for (int i = 0; i < length; ++i) {
		arr[i] = random_float();
	}
}

__global__ void bitonic_sort_step(float *dev_values, int j, int k)
{
	/* Сортировка i и ixj */
	unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;
	unsigned int ixj = i^j;

	/* Нити с наименьшими идентификаторами сортируют массив. */
	if (ixj > i) 
	{
		if (i & k == 0) 
		{
		/* Сортировка по возрастанию */
			if (dev_values[i] > dev_values[ixj]) 
			{
			/* обмен(i,ixj) */
				float temp = dev_values[i];
				dev_values[i] = dev_values[ixj];
				dev_values[ixj] = temp;
			}
		}
		if (i & k != 0) 
		{
			/* Сортировка по убыванию */
			if (dev_values[i] < dev_values[ixj]) 
			{
				/* обмен(i,ixj); */
				float temp = dev_values[i];
				dev_values[i] = dev_values[ixj];
				dev_values[ixj] = temp;
			}
		}
	}
}

/**
* Битонная сортировка на CUDA.
*/
void bitonic_sort(float *values)
{
	size_t size = NUM_VALS * sizeof(float);
	float *dev_values;
	hipMalloc((void**)&dev_values, size);
	hipMemcpy(dev_values, values, size, hipMemcpyHostToDevice);

	dim3 blocks(BLOCKS, 1); /* Количество блоков */
	dim3 threads(THREADS,1); /* Количество тредов */

	/* Основной шаг выполнения сортировки */
	for (int k = 2; k <= NUM_VALS; k <<= 1) 
	{
		/* Второстепенный шаг выполнения сортировки */
		for (int j = k>>1; j > 0; j = j>>1) 
		{
			bitonic_sort_step<<<blocks, threads>>>(dev_values, j, k);
		}
	}
	hipMemcpy(values, dev_values, size, hipMemcpyDeviceToHost);
	hipFree(dev_values);
}

int main(void)
{
	clock_t start, stop;
	float *values = (float*) malloc(NUM_VALS * sizeof(float));
	array_fill(values, NUM_VALS);
	start = clock();
	bitonic_sort(values);
	stop = clock();
	print_elapsed(start, stop);
}